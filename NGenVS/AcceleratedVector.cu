#include "hip/hip_runtime.h"
#include "AcceleratedVector.h"

#include <stdio.h>


///
//Allocates an accelerated vector on the GPU
//
//Parameters:
//	dim: the dimension of the vector to allocate
//
//Returns:
//	A pointer to an accelerated vector
AcceleratedVector* AcceleratedVector_Allocate()
{
	AcceleratedVector* aVec = (AcceleratedVector*)malloc(sizeof(AcceleratedVector));
	return aVec;

}

///
// Initializes an accelerated vector to match another vector of the same dimension
//
//Parameters:
//	aVec: 
void AcceleratedVector_Initialize(AcceleratedVector* aVec, const int dim)
{
	aVec->dimension = dim;
	hipMalloc((void**)&aVec->d_components, sizeof(float)* dim);
}

///
//Frees an accelerated vector
//
//Parameters:
//	aVec: Pointer to the accelerated vector to free
void AcceleratedVector_Free(AcceleratedVector* aVec)
{
	hipFree(aVec->d_components);
	free(aVec);
}

///
//Copies the contents of a vector to an accelerated vector of the same or larger dimension
//
//Parameters:
//	dest: A pointer to The accelerated vector to copy to
//	src: A pointer to the vector to copy from
void AcceleratedVector_CopyVector(AcceleratedVector* dest, const Vector* src)
{
	hipMemcpy(dest->d_components, src->components, sizeof(float)* src->dimension, hipMemcpyHostToDevice);
}


///
//Copies the contents from multiple vectors into a single concatenated accelerated vector
//The accelerated vector must have a dimension of at least the sum of the vectors dimension
//
//Parameters:
//	dest: A pointer to the accelerated vector to copy the concatenated contents to
//	srcs: An array of pointers to vectors to copy the contents from
//	dim: The dimension of all source vectors
//	numVectors: The number of source vectors
void AcceleratedVector_CopyVectors(AcceleratedVector* dest, const Vector** srcs, const unsigned int dim, const unsigned int numVectors)
{
	for (int i = 0; i < numVectors; i++)
	{
		hipMemcpy(dest->d_components + (i * dim), srcs[i]->components, sizeof(float)* dim, hipMemcpyHostToDevice);
	}
}

///
//Copies the contents of an accelerated vector to a vector
//Or pastes the contents of an acceleratedVector to a vector...
//Probably not the best name for this function but whatever.
//
//Parameters:
//	dest: The vector to paste the contents into
//	src: The vector to copy the contents of
void AcceleratedVector_PasteVector(Vector* dest, const AcceleratedVector* src)
{
	hipMemcpy(dest->components, src->d_components, sizeof(float)* src->dimension, hipMemcpyDeviceToHost);
}

///
//Pastes the contents of an accelerated vector where it's contents are the concatenated contents of various vectors
//Into an array of vectors
//
//Parameters:
//	dest: An array of vectors to paste the contents to
//	src: An accelerated vector to copy the contents from
//	dim: The dimension of each vector
//	numVectors: The number of vectors being pasted.
void AcceleratedVector_PasteVectors(Vector** dest, const AcceleratedVector* src, const unsigned int dim, const unsigned int numVectors)
{
	for (int i = 0; i < numVectors; i++)
	{
		hipMemcpy(dest[i]->components, src->d_components + (i * dim), sizeof(float)* dim, hipMemcpyDeviceToHost);
	}
}

///
//Increments a vector by another vector on the GPU
//Calculates possible optimal block/thread sizes then
//Calls Vector_AcceleratedIncrement
//
//Parameters:
//	d_dest: A device pointer to an array of floats representing the vector
//	d_src: A device pointer to an array of floats representing the incrementor
//	dim: The dimension of the vectors
void AcceleratedVector_LaunchIncrement(float* d_dest, const float* d_src, unsigned int dimension)
{
	int blockSize;
	int gridSize;
	int minBlockSize;
	//Get the max potential block size
	hipOccupancyMaxPotentialBlockSize(&minBlockSize, &blockSize, AcceleratedVector_Increment, 0, dimension);
	//Calculate the grid size
	gridSize = (dimension + blockSize - 1) / blockSize;

	//Call kernel
	AcceleratedVector_Increment << <gridSize, blockSize >> >(d_dest, d_src, dimension);
}

///
//Increments a vector by another vector on the GPU
//
//Parameters:
//	<<< Grid Size , Block Size >>>
//	d_dest: A device pointer to an array of floats representing the vector
//	d_src: A device pointer to an array of floats representing the incrementor
//	dim: The dimension of the vectors
__global__ void AcceleratedVector_Increment(float* dest, const float* src, const unsigned int dim)
{
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < dim)
	{
		dest[index] += src[index];
	}
}

///
//Adds two vectors storing the result in a third
//Calculates possible optimal grid/block sizes then calls
//Vector_AcceleratedAdd
//
//Parameters:
//	d_dest: a device pointer to an array of floats to store the sum vector
//	d_vec1: a device pointer to an array of floats representing the first addend vector
//	d_vec2: a device pointer to an array of floats representing the second addend vector
//	dim: the dimension of the vectors
void AcceleratedVector_LaunchAdd(float* d_dest, const float* d_vec1, const float* d_vec2, unsigned int dim)
{
	int blockSize;
	int gridSize;
	int minBlockSize;
	//Calculate potential optimal block / grid size
	hipOccupancyMaxPotentialBlockSize(&minBlockSize, &blockSize, AcceleratedVector_Add, 0, dim);
	gridSize = (dim + blockSize - 1) / blockSize;

	//Call accelerated add
	AcceleratedVector_Add << <gridSize, blockSize >> >(d_dest, d_vec1, d_vec2, dim);

}
///
//Adds two vectors storing the result in a third on the GPU
//
//Parameters:
//	<<< Grid Size , Block Size >>>
//	Grid Size: the number of blocks to execute in parallel
//	Block Size: The number of threads each block can have
//	d_dest: a device pointer to an array of floats to store the sum vector
//	d_vec1: a device pointer to an array of floats representing the first addend vector
//	d_vec2: a device pointer to an array of floats representing the second addend vector
//	dim: the dimension of the vectors
__global__ void AcceleratedVector_Add(float* d_dest, const float* d_vec1, const float* d_vec2, unsigned int dim)
{
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < dim)
	{
		d_dest[index] = d_vec1[index] + d_vec2[index];
	}
}

///
//Adds a concatenated array of vectors storing the sum in another vector on the GPU
//Calls AcceleratedVector_AddAll
//
//Parameters:
//	d_dest: A device pointer to an array of floats representing the vector to store the sum
//	d_srcs: A device pointer to an array of floats representing the concatenated contents of the vectors to sum
//	dim: The dimension of the vectors
//	numVectors: Thenumber of vectors to sum
void AcceleratedVector_LaunchAddAll(float* d_dest, const float* d_srcs, int dim, int numVectors)
{
	AcceleratedVector_AddAll << <dim, numVectors / 2, ceilf(numVectors / 2.0f) * sizeof(float) >> >(d_dest, d_srcs, dim, numVectors);
}
///
//Adds a concatenated array of vectors storing the sum in another vector on the GPU
//
//Parameters:
//	<<< Grid Size, Block Size >>>
//	grid size:	The dimension of the vectors (One block will compute each index)
//	block size: The number of vectors to add (One thread per vector)
//	d_dest: A device pointer to an array of floats representing the vector to store the sum
//	d_srcs: A device pointer to an array of floats representing the concatenated contents of the vectors to sum
//	dim: The dimension of the vectors
//	numVectors: Thenumber of vectors to sum
__global__ void AcceleratedVector_AddAll(float* d_dest, const float* d_srcs, int dim, int numVectors)
{
	int sumSize = ceilf(numVectors / 2.0f);
	extern __shared__ float sum[];	//Array to be used for reduction addition
	//Dimension of above array is passed in from AcceleratedVector_LaunchAddAll using special optional third parameter
	//In triple angle brackets.

	//let the threadID be the vector this thread is responsible for summing with the vector next to it.
	//Let blockIdx.x be the element this thread is resposible for summing. 
	if (threadIdx.x < (numVectors / 2))	//Do not perform any operations if the thread ID is outside the range of vectors/2.
	{
		int index1 = blockIdx.x + (threadIdx.x * 2) * dim;	//Each thread will take element blockIdx.x of vector threadIdx
		int index2 = blockIdx.x + (((threadIdx.x * 2) + 1) * dim);	//And element blockIdx.x of vector threadIdx + 1
		//Thread 0 of block x compute the sum of the x element of the first two vectors storing the result in sum[0]
		//Threads 1 of block x compute the sum of the x element of the second and third vectors storing the result in sum[1]
		//Thread y of block x computes the sum of the x element of the 2y and 2y+1 vectors storing the result in sum[y]
		sum[threadIdx.x] = d_srcs[index1] + d_srcs[index2];

		if (numVectors % 2 == 1)	//If there is an extra vector which was not included in the sum
		{
			//0th thread will Set sum[sumSize-1] = missing index
			if (threadIdx.x == 0)
			{
				sum[sumSize - 1] = d_srcs[blockIdx.x + ((numVectors - 1) * dim)];
			}

		}
	}

	//Fill sum before going any further
	__syncthreads();

	//Begin reduction
	AcceleratedVector_dReduceArray(sum, sumSize);

	//Wait for all threads here
	__syncthreads();



	if (threadIdx.x == 0)
	{
		//Wrte final sum to each index of d_dest
		d_dest[blockIdx.x] = sum[0];
	}


}

///
//Decrements one vector by another on the GPU
//First calculates possible optimal block and grid sizes
//Then calls Vector_AcceleratedDecrement
//
//Parameters:
//	d_dest: A device pointer to an array of floats representing the vector being decremented
//	d_src: A device pointer to an array of floats representing the vector to increment by
//	dim: The dimension of the two vectors
void AcceleratedVector_LaunchDecrement(float* d_dest, const float* d_src, unsigned int dim)
{
	int blockSize;
	int gridSize;
	int minBlockSize;

	hipOccupancyMaxPotentialBlockSize(&minBlockSize, &blockSize, AcceleratedVector_Decrement, 0, dim);
	gridSize = (dim + blockSize - 1) / blockSize;

	AcceleratedVector_Decrement << <gridSize, blockSize >> >(d_dest, d_src, dim);
}
///
//Decrements one vector by another on the GPU
//
//Parameters:
//	<<< Grid Size , Block Size >>>
//	Grid Size: The number of blocks to execute in parallel
//	Block Size: The nuber of threads each block can have
//	d_dest: A device pointer to an array of floats representing the vector being decremented
//	d_src: A device pointer to an array of floats representing the vector to increment by
//	dim: The dimension of the two vectors
__global__ void AcceleratedVector_Decrement(float* d_dest, const float* d_src, unsigned int dim)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < dim)
	{
		d_dest[index] -= d_src[index];
	}
}


///
//calculates vec1 - vec2 on GPU
//Calculates possible optimal block/thread sizes then
//Calls Vector_AcceleratedIncrement
//
//Parameters:
//	<<< Grid Size , Block Size >>>
//	Grid Size: the number of blocks to execute in parallel
//	Block Size: The number of threads each block can have
//	d_dest: A device pointer to an array of floats where the result can be stored
//	d_vec1: A device pointer to an array of floats representing the Left hand side vector operand
//	d_vec2: A device pointer to an array of floats representing the Right had side vector operand
//	dim: The dimension of the vectors
void AcceleratedVector_LaunchSubtract(float* d_dest, const float* d_vec1, const float* d_vec2, unsigned int dim)
{
	int blockSize;
	int gridSize;
	int minBlockSize;
	//Get the max potential block size
	hipOccupancyMaxPotentialBlockSize(&minBlockSize, &blockSize, AcceleratedVector_Subtract, 0, dim);
	//Calculate the grid size
	gridSize = (dim + blockSize - 1) / blockSize;

	AcceleratedVector_Subtract << <gridSize, blockSize >> >(d_dest, d_vec1, d_vec2, dim);

}
///
//Let d_dest, d_vec1, and d_vec2 be dim dimension vectors
//Computes d_vec1 - d_vec2 and stores the result in d_dest
//
//Parameters:
//	<<< Grid Size , Block Size >>>
//	Grid Size: the number of blocks to execute in parallel
//	Block Size: The number of threads each block can have
//	d_dest: A device pointer to an array of floats where the result can be stored
//	d_vec1: A device pointer to an array of floats representing the Left hand side vector operand
//	d_vec2: A device pointer to an array of floats representing the Right had side vector operand
//	dim: The dimension of the vectors
__global__ void AcceleratedVector_Subtract(float* d_dest, const float* d_vec1, const float* d_vec2, unsigned int dim)
{
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < dim)
	{
		d_dest[index] = d_vec1[index] - d_vec2[index];
	}
}

///
//Gets the scalar product of a vector with a scalar on the GPU
//Computes a possible optimal gridsize and blocksize then calls AcceleratedVector_GetScalarProduct
//
//Parameters:
//	d_dest: The destination of the scaled vector
//	d_vec1: The vector to scale
//	scalar: The scalar to scale the vector by
//	dim: The dimension of the vector getting scaled
void AcceleratedVector_LaunchGetScalarProduct(float* d_dest, const float* d_vec1, const float scalar, unsigned int dim)
{
	int blockSize;
	int gridSize;
	int minBlockSize;
	//Get the max potential block size
	hipOccupancyMaxPotentialBlockSize(&minBlockSize, &blockSize, AcceleratedVector_Subtract, 0, dim);
	//Calculate the grid size
	gridSize = (dim + blockSize - 1) / blockSize;

	AcceleratedVector_GetScalarProduct << <gridSize, blockSize >> >(d_dest, d_vec1, scalar, dim);
}

///
//Gets the scalar product of a vector with a scalar on the GPU
//
//Parameters:
//	<<<Grid Size, Block Size>>>
//	Grid Size: The number of blocks to execute in parallel
//	BlockSize: The number of threads to execute in parallel
//	d_dest: The destination of the scaled vector
//	d_vec1: The vector to scale
//	scalar: The scalar to scale the vector by
//	dim: The dimension of the vector getting scaled
__global__ void AcceleratedVector_GetScalarProduct(float* d_dest, const float* d_vec1, const float scalar, unsigned int dim)
{
	float* d_blockDest = d_dest + (blockIdx.x * blockDim.x);
	const float* d_blockVec1 = d_vec1 + (blockIdx.x * blockDim.x);
	//AcceleratedVector_dGetScalarProduct(d_dest, d_vec1, scalar, dim);
	AcceleratedVector_dGetScalarProduct(d_blockDest, d_blockVec1, scalar, dim);
}

///
//Gets the scalar product of a vector with a scalar on the GPU
//
//Parameters:
//	d_dest: The destination of the scaled vector
//	d_Vec1: The vector to scale
//	scalar: The scalar to scale the vector by
//	dim: The dimension of the vector getting scaled
__device__ void AcceleratedVector_dGetScalarProduct(float* d_dest, const float* d_vec1, const float scalar, unsigned int dim)
{
	//unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	/*
	if (index < dim)
	{
		d_dest[index] = d_vec1[index] * scalar;
	}
	*/

	if (threadIdx.x < dim)
	{
		d_dest[threadIdx.x] = d_vec1[threadIdx.x] * scalar;
	}
}

///
//Scales an accelerated vector by a scalar value on the GPU
//Calculates possible optimal block & grid dimensions
//Then calls AcceleratedVector_Scale
//
//Parametrs:
//	d_dest: A device pointer to the vector to scale
//	scalar: The scalar value toscale the vector by
//	dim: The dimension of the vector ebing scale
void AcceleratedVector_LaunchScale(float* d_dest, const float scalar, const unsigned int dim)
{
	int blockSize;
	int gridSize;
	int minBlockSize;
	//Get the max potential block size
	hipOccupancyMaxPotentialBlockSize(&minBlockSize, &blockSize, AcceleratedVector_Subtract, 0, dim);
	//Calculate the grid size
	gridSize = (dim + blockSize - 1) / blockSize;

	AcceleratedVector_Scale << <gridSize, blockSize >> >(d_dest, scalar, dim);

}

///
//Scales an Acceeereated vector by a scalar value on the GPU
//Calls AcceleratedVector_dScale
//
//PArameters:
//	<<< Grid Size, Block Size>>>
//	Grid Size: The number of blocks to execute in parallel
//	Block size: The number of threads each block will execute in parallel
//	d_Dest: A device pointer to the vector to scale
//	scalar: The scalar to scale the vector by
//	dim: The dimension of the vector being scaled
__global__ void AcceleratedVector_Scale(float* d_dest, const float scalar, const unsigned int dim)
{
	AcceleratedVector_dScale(d_dest, scalar, dim);
}

///
//Scales an accelerated vector by a scalar value on the GPU
//
//Parameters:
//	d_dest: A device pointer to the vector to scale
//	scalar: The scalar to scae the vector by
//	dim: THe dimension of the vector being scaled
__device__ void AcceleratedVector_dScale(float* d_dest, const float scalar, const unsigned int dim)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < dim)
	{
		d_dest[index] *= scalar;
	}
}

///
//Scales multiple accelerated vectors at once by different scale values on the GPU
//Determines Block & Grid sizes then calls AcceleratedVector_ScaleAll
//
//PArameters:
//	d_dests: A device pointer to an array of floats representing the concatenated components of the vectors being scaled
//	scalars: A device pointer to an array of scalars respective to the array of vectors that they scale
//	dim: The dimension of each vector
//	numVectors: The number of vectors to scale
void AcceleratedVector_LaunchScaleAll(float* d_dests, const float* d_scalars, const unsigned int dim, const unsigned int numVectors)
{
	AcceleratedVector_ScaleAll <<<numVectors, dim >>>(d_dests, d_scalars, dim, numVectors);

}

///
//Scales multiple accelerated vectors at once by different scale values on the GPU
//
//PArameters:
//	<<< Grid Size, Block Size>>>
//	Grid Size: The number of blocks to execute in parallel
//	Block size: The number of threads each block will execute in parallel
//	d_dests: A device pointer to an array of floats representing the concatenated components of the vectors being scaled
//	scalars: A device pointer to an array of scalars respective to the array of vectors that they scale
//	dim: The dimension of each vector
//	numVectors: The number of vectors to scale
__global__ void AcceleratedVector_ScaleAll(float* d_dests, const float* d_scalars, const unsigned int dim, const unsigned int numVectors)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < dim * numVectors)
	{
		d_dests[index] *= d_scalars[blockIdx.x];
	}
}

///
//Calculates the magnitude of a vector on the GPU
//Determines block & grid sizes then calles AcceleratedVector_Magnitude
//
//Parameters:
//	d_mag: A device pointer to store the magnitude in
//	d_vector: A device pointer to an array of floats representing the components of the vector to calculate the magnitude of
//	dim: The dimension of the vector
void AcceleratedVector_LaunchMagnitude(float* d_mag, const float* d_vector, const unsigned int dim)
{
	AcceleratedVector_Magnitude << <1, dim, dim * sizeof(float) >> >(d_mag, d_vector, dim);

}

///
//Calculates the magnitude of a vector on the GPU
//
//Parameters:
//	<<<GridSize, Block Size, Dynamic Allocation>>>
//	Grid Size: The number of blocks to run in parallel (Should be 1 for this algorithm)
//	Block Size: The number of threads each block will run in parallel. Should be equal to the dimension of the vector * the size of each component.
//	Dynamic Allocation: This algorithm needs to allocate memory equal to the 
//	d_mag: A device pointer to store the magnitude in
//	d_vector: A device pointer to an array of floats representing the components of the vector to calculate the magnitude of
//	dim: The dimension of the vector
__global__ void AcceleratedVector_Magnitude(float* d_mag, const float* d_vector, const unsigned int dim)
{
	//AcceleratedVector_dMagnitude(d_mag, d_vector, dim);

	extern __shared__ float squares[];	//Array to be used for reduction addition
	
	AcceleratedVector_dMagnitude(d_mag, d_vector, dim, squares);

	/*
	if (threadIdx.x < dim)
	{
		//Fill squares with the square of each component
		squares[threadIdx.x] = powf(d_vector[threadIdx.x], 2.0f);
	}

	//Wait for all threads to complete
	__syncthreads();

	//Begin reduction
	AcceleratedVector_dReduceArray(squares, dim);

	//Wait for all threads to complete
	__syncthreads();

	if (threadIdx.x == 0)
	{
		d_mag[0] = sqrtf(squares[0]);
	}
	*/
}


///
//Calculates the magnitude of a Vector on the GPU
//This function is only callable from the device
//Each thread is in charge of squaring it's respective element. Then reduction
// is used to find the sum and that is returned.
//
//Parameters:
//	d_mag: A device pointer to a float to store the magnitude
//	d_vector: A device pointer to an array of floats representing the vector to get he magnitude of
//	dim: The dimension of the vector tg et the magnitude from
//	d_sharedStorage: A device pointer to shared memory with dim * sizeof(vector element) space. This is to perform reduction.
__device__ void AcceleratedVector_dMagnitude(float* d_mag, const float* d_vector, const unsigned int dim, float* d_sharedStorage)
{
	/*
	if (threadIdx.x < dim)
	{
		//Fill squares with the square of each component
		d_sharedStorage[threadIdx.x] = powf(d_vector[threadIdx.x], 2.0f);
	}



	//Wait for all threads to complete
	__syncthreads();

	//Begin array reduction
	AcceleratedVector_dReduceArray(d_sharedStorage, dim);

	//Wait for all threads to complete
	__syncthreads();

	if (threadIdx.x == 0)
	{
		d_mag[0] = sqrtf(d_sharedStorage[0]);
	}*/
	AcceleratedVector_dMagnitudeSq(d_mag, d_vector, dim, d_sharedStorage);
	
	if (threadIdx.x == 0)
	{
		d_mag[0] = sqrtf(d_mag[0]);
	}

}

///
//Calculates the magnitude squared of a vector on the GPU
//This function is only callable from the device
//
//Parameters:
//	d_mag: A device pointer to a float to store the magnitude
//	d_vector: A device pointer to an array of floats representing the vector to get he magnitude of
//	dim: The dimension of the vector tg et the magnitude from
//	d_sharedStorage: A device pointer to shared memory with dim * sizeof(vector element) space
__device__ void AcceleratedVector_dMagnitudeSq(float* d_mag, const float* d_vector, const unsigned int dim, float* d_sharedStorage)
{
	//int index = threadIdx.x + blockIdx.x + blockDim.x;
	if (threadIdx.x < dim)
	{
		//Fill squares with the square of each component
		d_sharedStorage[threadIdx.x] = powf(d_vector[threadIdx.x], 2.0f);
	}



	//Wait for all threads to complete
	__syncthreads();

	//Begin array reduction
	AcceleratedVector_dReduceArray(d_sharedStorage, dim);

	//Wait for all threads to complete
	__syncthreads();

	if (threadIdx.x == 0)
	{
		d_mag[0] = d_sharedStorage[0];
	}
}

///
//Gets a Normalized a vector the GPU
//First determines gridsize and blocksize then calls AcceleratedVector_Normalize
//
//Parameters:
//	d_dest: Device pointer to an array to store the components of the normalized src
//	d_src: Device pointer to an array representing the components of the vector to normalize
//	dim: The dimension of th vector
void AcceleratedVector_LaunchGetNormalize(float* d_dest, const float* d_src, const unsigned int dim)
{
	AcceleratedVector_GetNormalize << <1, dim, dim * sizeof(float) >> >(d_dest, d_src, dim);
}

///
//Gets a Normalized a vector the GPU
//
//Parameters:
//	<<<Grid Size, Block Size>>>
//	Grid Size: The number of blocks to run in parallel. Should be 1 for this algorithm.
//	Block Size: The number of threads each block should run in parallel. Should be equal to the dimension of the vector being normalized
//	d_dest: A device pointer to an array of floats to store the components of the normalized vector
//	d_src: Adevice pointer to an array of floats representing the components of the vector to normalize
//	dim: The dimension of the vector to normalize
__global__ void AcceleratedVector_GetNormalize(float* d_dest, const float* d_src, const unsigned int dim)
{
	extern __shared__ float reductionStorage[];	//Used for performing reduction in AcceleratedVector_dMagnitude
	__shared__ float magnitude;

	AcceleratedVector_dMagnitude(&magnitude, d_src, dim, reductionStorage);

	//Wait for magnitude
	__syncthreads();

	if (magnitude != 0)
	{
		AcceleratedVector_dGetScalarProduct(d_dest, d_src, 1.0f / magnitude, dim);
	}
}

///
//Normalizes a vector on the GPU
//First determines gridsize and blocksize then calls AcceleratedVector_Normalize
//
//PArameters:
//	d_vec: A device pointer to an array of floats representing the components to normalize
//	dim: The dimension of the vector
void AcceleratedVector_LaunchNormalize(float* d_vec, const unsigned int dim)
{
	AcceleratedVector_Normalize << <1, dim, dim * sizeof(float) >> >(d_vec, dim);


}

///
//Normalizes a vector on the GPU
//
//PArameters:
//	<<<Grid Size, Block Size, Dynamic Allocation>>>
//	Grid Size: The number of blocks to execute in parallel. Should be 1 for this algorithm
//	Block Size: The number of threads to run in parallel. Should be equal to the vector dimension for this algorithm
//	Dynamic Allocation: Algorithm needs additional memory on GPU to run equal to dim * sizeof(vector element)
//	d_vec: A device pointer to an array of floats representing the components to normalize
//	dim: The dimension of the vector
__global__ void AcceleratedVector_Normalize(float* d_vec, const unsigned int dim)
{
	extern __shared__ float reductionStorage[];	//Used for performing reduction in AcceleratedVector_dMagnitude
	__shared__ float magnitude;

	AcceleratedVector_dMagnitude(&magnitude, d_vec, dim, reductionStorage);


	//Wait for magnitude from thread0
	__syncthreads();

	if (magnitude != 0)
	{
		//Scale vector by inverse magnitude
		AcceleratedVector_dScale(d_vec, 1.0f / magnitude, dim);
	}
	
}


///
//Computes the dot product of two vectors on the GPU
//
//Parameters:
//	d_dest: Pointer to a float on the GPU to store the result of the dot product
//	d_vec1: a device pointer to an array of floats representing the first vector being dotted
//	d_vec2: a device pointer to an array of floats representing the second vector to be dotted
//	dim: The dimension of the vectors being dotted
void AcceleratedVector_LaunchDotProduct(float* d_dest, const float* d_vec1, const float* d_vec2, unsigned int dim)
{
	AcceleratedVector_DotProduct << <1, dim, dim * sizeof(float) >> >(d_dest, d_vec1, d_vec2, dim);
}


///
//Computes the dot product of two vectors on the GPU
//
//Parameters:
//	<<< Grid Size, Block Size >>>
//	Grid Size: Single dot product must be launched in one block.
//	Block Size: Number of threads to give the active block. Equal to the dimension of vectors being dotted.
//	d_dest: Pointer to a float on the GPU to store the result of the dot product
//	d_vec1: a device pointer to an array of floats representing the first vector being dotted
//	d_vec2: a device pointer to an array of floats representing the second vector to be dotted
//	dim: The dimension of the vectors being dotted
__global__ void AcceleratedVector_DotProduct(float* d_dest, const float* d_vec1, const float* d_vec2, unsigned int dim)
{
	extern __shared__ float products[];	//Array to be used for reduction addition

	AcceleratedVector_dDotProduct(d_dest, d_vec1, d_vec2, dim, products);

	/*
	if (threadIdx.x < dim)
	{
		products[threadIdx.x] = d_vec1[threadIdx.x] * d_vec2[threadIdx.x];

		__syncthreads();

		//Reduce array of products
		AcceleratedVector_dReduceArray(products, dim);

		if (threadIdx.x == 0)
		{
			d_dest[0] = products[0];
		}
	}
	*/

}

///
//Computes the dot product of two vectors on the GPU
//
//Parameters:
//	d_dest: Pointer to a float on the GPU to store the result of the dot product
//	d_vec1: a device pointer to an array of floats representing the first vector being dotted
//	d_vec2: a device pointer to an array of floats representing the second vector to be dotted
//	dim: The dimension of the vectors being dotted
//	reductionStorage: A device pointer to an array of floats with dim components to be used for reduction
__device__ void AcceleratedVector_dDotProduct(float* d_dest, const float* d_vec1, const float* d_vec2, unsigned int dim, float* reductionStorage)
{
	//int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (threadIdx.x < dim)
	{
		//reductionStorage[threadIdx.x] = d_vec1[index] * d_vec2[index];
		reductionStorage[threadIdx.x] = d_vec1[threadIdx.x] * d_vec2[threadIdx.x];
	}
	//Wait for all threads to complete
	__syncthreads();

	//Reduce array to get sum
	AcceleratedVector_dReduceArray(reductionStorage, dim);

	if (threadIdx.x == 0)
	{
		d_dest[0] = reductionStorage[0];
	}


}

///
//Computes the dot product of multiple vectors on the GPU
//
//Parameters:
//	d_dest: A device pointer to an array of floats to store the respective dot product results
//	d_vecs1: A device pointer to an array of floats representing numVectors vectors of dimension dim as the LHS vectors
//	d_vecs2: A device pointer to an array of floats represnting numVectors vectors of dimension dim as the RHS vectors
//	dim: The dimension of the vectors being dotted
//	numVectors: number of vectors in each vector component array
void AcceleratedVector_LaunchDotProductAll(float* d_dest, const float* d_vecs1, const float* d_vecs2, unsigned int dim, unsigned int numVectors)
{
	AcceleratedVector_DotProductAll << <numVectors, dim, dim * sizeof(float) >> >(d_dest, d_vecs1, d_vecs2, dim, numVectors);
}

///
//Computes the dot product of multiple vectors on the GPU
//
//Parameters:
//	<<< Grid Size , Block size >>>
//	Grid Size : The amount of dot products being performed, numVectors
//	Block Size: The amount of threads to run per block, in this case, equal to the dimension of the vectors
//	d_dest: A device pointer to an array of floats to store the respective dot product results
//	d_vecs1: A device pointer to an array of floats representing a contiguous array of multiple sets of vector components representing the LHS vectors
//	d_vecs2: A device pointer to an array of floats representing a contiguous array of multiple sets of vector components representing the RHS vectors
//	dim: The dimension of the vectors being dotted
//	numVectors: The number of vectors in each vector component array
__global__ void AcceleratedVector_DotProductAll(float* d_dest, const float* d_vecs1, const float* d_vecs2, unsigned int dim, unsigned int numVectors)
{
	extern __shared__  float products[];

	float* d_blockDest = d_dest + (blockIdx.x);
	const float* d_blockVecs1 = d_vecs1 + (blockIdx.x * blockDim.x);
	const float* d_blockVecs2 = d_vecs2 + (blockIdx.x * blockDim.x);

	//AcceleratedVector_dDotProductAll(d_dest, d_vecs1, d_vecs2, dim, numVectors, products);
	AcceleratedVector_dDotProductAll(d_blockDest, d_blockVecs1, d_blockVecs2, dim, numVectors, products);
}

///
//Computes the dotprduct of multiple vectors on the GPU
//
//Parameters:
//	d_dest: A device pointer to the proper index in an array of floats to store respective dot product result
//	d_vecs1: A device pointer to an array of floats representing the concatenation of all LHS vecto components involved in dot products
//	d_vecs2: A device pointer to an array of floats representing the concatenated components of all RHS vectors involved in dot products
//	dim: The dimension of the vectors being dotted
//	numVectors: The number of vectors being
//	d_reductionStorage: A device pointer to an array of floats of length sizeof(VectorElement) * dim
__device__ void AcceleratedVector_dDotProductAll(float* d_dest, const float* d_vecs1, const float* d_vecs2, unsigned int dim, unsigned int numVectors, float* d_reductionStorage)
{
	//Calculate the index this thread will get the product from the LHS and RHS
	//int index = threadIdx.x + blockIdx.x * blockDim.x;

	//If the thread ID is within the dimension of the vector
	if (threadIdx.x < dim)
	{

		//Store the product of LHS[index] * RHS[index] in product at index corresponding to threadID
		//d_reductionStorage[threadIdx.x] = d_vecs1[index] * d_vecs2[index];
		d_reductionStorage[threadIdx.x] = d_vecs1[threadIdx.x] * d_vecs2[threadIdx.x];
	}
	//Wait for threads to finish computing all of products
	__syncthreads();

	//Use regression to find the sum of the products
	AcceleratedVector_dReduceArray(d_reductionStorage, dim);

	if (threadIdx.x == 0)
	{
		d_dest[0] = d_reductionStorage[0];
	}
}

///
//Computes the dot product of a set of vectors with another vector on the GPU.
//Stores the result in an array containing the result of each respective dot product
//Determines the grid size and block size then calls AcceleratedVector_DotProductAllWith
//
//Parameters:
//	d_dest: A device pointer to an array of floats containing the respective dot product results
//	d_vec1: A device pointer to an array of floats contaning the components of the constant vector involved in the dot products
//	d_vecs2: A device pointer to an array of floats containing the contiguous components of the set of vectors each being dotted with d_vec1
//	dim: The dimension of the vectors
//	numVectors: The number of vectors involved in the dot product
void AcceleratedVector_LaunchDotProductAllWith(float* d_dest, const float* d_vec1, const float* d_vecs2, unsigned int dim, unsigned int numVectors)
{
	AcceleratedVector_DotProductAllWith << <numVectors, dim, sizeof(float)* dim >> >(d_dest, d_vec1, d_vecs2, dim, numVectors);
}

///
//Computes the dot product of a set of vectors with another vector on the GPU.
//Stores the result in an array containing the result of each respective dot product
//Determines the grid size and block size then calls AcceleratedVector_DotProductAllWith
//
//Parameters:
//	<<<Grid Size, Block Size, Dynamic Allocation>>>
//	Grid Size: The number of blocks to execute in parallel. Should be equal to the number of vectors in this case.
//	Block Size: The number of threads each block should execute in parallel. Should be equal to the dimension of the vector.
//	Dynamic Allocation: Array of memory to dynamically allocate for this function. This function requires sizeof(vectorElement) * dim extra space per block.
//	d_dest: A device pointer to an array of floats containing the respective dot product results
//	d_vec1: A device pointer to an array of floats contaning the components of the constant vector involved in the dot products
//	d_vecs2: A device pointer to an array of floats containing the contiguous components of the set of vectors each being dotted with d_vec1
//	dim: The dimension of the vectors
//	numVectors: The number of vectors involved in the dot product
__global__ void AcceleratedVector_DotProductAllWith(float* d_dest, const float* d_vec1, const float* d_vecs2, unsigned int dim, unsigned int numVectors)
{
	extern __shared__ float products[];

	//Get products of each element
	if (threadIdx.x < dim)
	{
		products[threadIdx.x] = d_vec1[threadIdx.x] * d_vecs2[threadIdx.x + blockIdx.x * blockDim.x];
	}

	__syncthreads();

	//Use reduction to get sum of products
	AcceleratedVector_dReduceArray(products, dim);

	__syncthreads();

	if (threadIdx.x == 0)
	{
		d_dest[blockIdx.x] = products[0];
	}
}

///
//Projects d_vec1 onto d_vec2 storing the projection vector in d_dest on the GPU
//Detemrines the grid dimension and block dimension then calls AcceleratedVector_GetProjection
//
//Parameters:
//	d_dest: A device pointer to an array of floats to store the result of the projection
//	d_vec1: A device pointer to an array of floats representing the components of The vector being projected
//	d_vec2: A device pointer to an array of floats containing the components of the vector being projected onto
//	dim: Te dimension of the vectors (The vectors dimension should match)
void AcceleratedVector_LaunchGetProjection(float* d_dest, const float* d_vec1, const float* d_vec2, const unsigned int dim)
{
	AcceleratedVector_GetProjection << <1, dim, sizeof(float)* dim >> >(d_dest, d_vec1, d_vec2, dim);

}

///
//Projects d_vec1 onto d_vec2 storing the projection vector in d_dest on the GPU
//
//Parameters:
//	<<<Grid Size, Block Size, Dynamic Allocation>>>
//	Grid Size: The amount of blocks to execute in parallel. Should be one for thisalgorithm
//	Block size: The aount of threads each block should execute in parallel. Should be equal to the dimension of the vectors being projected
//	Dynamic Allocation: The amount of extra data this algorithm will need to perform (Needs sizeof(VectorElement) * dim bytes)
//	d_dest: A device pointer to an array of floats to store the result of the projection
//	d_vec1: A device pointer to an array of floats representing the components of The vector being projected
//	d_vec2: A device pointer to an array of floats containing the components of the vector being projected onto
//	dim: Te dimension of the vectors (The vectors dimension should match)
__global__ void AcceleratedVector_GetProjection(float* d_dest, const float* d_vec1, const float* d_vec2, const unsigned int dim)
{
	extern __shared__ float reductionStorage[];
	__shared__ float numerator;		//U dot V
	__shared__ float denominator;	//Mag(V) Squared


	AcceleratedVector_dDotProduct(&numerator, d_vec1, d_vec2, dim, reductionStorage);
	//Wait for all threads to complete
	__syncthreads();

	AcceleratedVector_dMagnitudeSq(&denominator, d_vec2, dim, reductionStorage);

	__syncthreads();
	if (threadIdx.x == 0)
	{
		numerator = numerator / denominator;
	}

	__syncthreads();

	AcceleratedVector_dGetScalarProduct(d_dest, d_vec2, numerator, dim);
}

///
//Projects d_vec1 onto d_vec2 changing d_vec1 to represent the projection vector on the GPU
//Detemrines the grid dimension and block dimension then calls AcceleratedVector_LaunchProject
//
//Parameters:
//	d_vec1: A device pointer to an array of floats representing the components of The vector being projected
//	d_vec2: A device pointer to an array of floats containing the components of the vector being projected onto
//	dim: Te dimension of the vectors (The vectors dimension should match
void AcceleratedVector_LaunchProject(float* d_vec1, const float* d_vec2, const unsigned int dim)
{
	AcceleratedVector_Project << <1, dim, sizeof(float)* dim >> >(d_vec1, d_vec2, dim);
}

///
//Projects d_vec1 onto d_vec2 on the GPU. Result is stored in d_vec1.
//
//Parameters:
//	<<<Grid Size, Block Size, Dynamic Allocation>>>
//	GridSize: The amount of blocks to execute in parallel (should be one for this algorithm)
//	blockSize: The amount of threads each block should execute in parallel (Should be equal to the dimension of the vectors)
//	Dynamic Allocation: The amount of extra data this algorithm will need to perform (Needs sizeof(VectorElement) * dim bytes)
//	d_vec1: A device pointer to an array of floats representing the components of The vector being projected
//	d_vec2: A device pointer to an array of floats containing the components of the vector being projected onto
//	dim: Te dimension of the vectors (The vectors dimension should match
__global__ void AcceleratedVector_Project(float* d_vec1, const float* d_vec2, const unsigned int dim)
{
	extern __shared__ float reductionStorage[];
	__shared__ float numerator;		//U dot V
	__shared__ float denominator;	//Mag(V) Squared


	AcceleratedVector_dDotProduct(&numerator, d_vec1, d_vec2, dim, reductionStorage);
	//Wait for all threads to complete
	__syncthreads();

	AcceleratedVector_dMagnitudeSq(&denominator, d_vec2, dim, reductionStorage);

	__syncthreads();
	if (threadIdx.x == 0)
	{
		numerator = numerator / denominator;
	}


	__syncthreads();

	AcceleratedVector_dGetScalarProduct(d_vec1, d_vec2, numerator, dim);
}

///
//Projects each vector to be projected onto the respective vector being projected onto, altering the LHS vector to hold the solution
//Determines the grid size and block size then calls AcceleratedVector_ProjectAll
//
//Parameters:
//	d_vecs1: A device pointer to an array of floats representing the concatenated components of each vector being projected.
//	d_vecs2: A device pointer to an array of floats representing the concatenated components of each vector being projected onto.
//	dim: The dimension of the vectors being projected
//	numVectors: The number of vectors being projected
void AcceleratedVector_LaunchProjectAll(float* d_vecs1, const float* d_vecs2, const unsigned int dim, const unsigned int numVectors)
{
	AcceleratedVector_ProjectAll<<<numVectors, dim, sizeof(float)* dim>>>(d_vecs1, d_vecs2, dim, numVectors);
}

///
//Projects each vector to be projected onto the respective vector being projected onto, altering the LHS vector to hold the solution
//Determines the grid size and block size then calls AcceleratedVector_ProjectAll
//
//Parameters:
//	<<<Grid Size, Block Size, Dynamic Allocation>>>
//	Grid Size: THe amount of blocks to execute in parallel. The number of vectors being projected
//	Block size: THe amount of threads each block can execute in parallel. The dimension of the vectors being projected
//	Dynamic Allocation: The aount of extra data this algorithm will need to perform. Needs sizeof(VectorElement) * dim bytes.
//	d_vecs1: A device pointer to an array of floats representing the concatenated components of each vector being projected.
//	d_vecs2: A device pointer to an array of floats representing the concatenated components of each vector being projected onto.
//	dim: The dimension of the vectors being projected
//	numVectors: The number of vectors being projected
__global__ void AcceleratedVector_ProjectAll(float* d_vecs1, const float* d_vecs2, const unsigned int dim, const unsigned int numVectors)
{
	extern __shared__ float d_reductionStorage[];
	__shared__ float numerator;
	__shared__ float denominator;

	__shared__ float* d_blockVecs1;
	__shared__ const float* d_blockVecs2;

	if (threadIdx.x == 0)
	{
		d_blockVecs1 = d_vecs1 + (blockIdx.x * blockDim.x);
		d_blockVecs2 = d_vecs2 + (blockIdx.x * blockDim.x);
	}
	
	__syncthreads();

	//AcceleratedVector_dDotProduct(&numerator, d_vecs1, d_vecs2, dim * numVectors, d_reductionStorage);
	AcceleratedVector_dDotProduct(&numerator, d_blockVecs1, d_blockVecs2, dim, d_reductionStorage);

	__syncthreads();

	//AcceleratedVector_dMagnitudeSq(&denominator, d_vecs2, dim * numVectors, d_reductionStorage);
	AcceleratedVector_dMagnitudeSq(&denominator, d_blockVecs2, dim, d_reductionStorage);

	__syncthreads();

	if (threadIdx.x == 0)
	{
		numerator = numerator / denominator;
	}


	__syncthreads();

	//AcceleratedVector_dGetScalarProduct(d_vecs1, d_vecs2, numerator, dim * numVectors);
	AcceleratedVector_dGetScalarProduct(d_blockVecs1, d_blockVecs2, numerator, dim);

}

///
//Uses reduction techniques to compute the sum of components in the array storing te sum in the first element
//There must be at least arrSize threads.
//
//Parameters:
//	arrToReduce: The array to reduce
//	arrSize: The size of the array
__device__ void AcceleratedVector_dReduceArray(float* arrToReduce, const unsigned int arrSize)
{
	for (int i = 1; i < arrSize; i *= 2)
	{
		//Add every other, then every 4, then every 8 ... indices together!
		if (threadIdx.x % (i * 2) == 0 && threadIdx.x + i < arrSize)
		{
			arrToReduce[threadIdx.x] += arrToReduce[threadIdx.x + i];
		}
		__syncthreads();
	}
}